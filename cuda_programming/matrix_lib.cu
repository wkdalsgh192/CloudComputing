#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define TILE_SIZE 16

__global__ void matrixMultiplyTiled(float *A, float *B, float *C, int N)
{

    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    // Each thread computes one element of the output matrix
    int row = blockIdx.y * TILE_SIZE + threadIdx.y; // global row index
    int col = blockIdx.x * TILE_SIZE + threadIdx.x; // global col index

    float sum = 0.0f;
    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; t++)
    {
         // load elements into shared memory if within bounds
        if (row < N && t * TILE_SIZE + threadIdx.x < N)
            tileA[threadIdx.y][threadIdx.x] = A[row * N + t * TILE_SIZE + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0f;

        if (t * TILE_SIZE + threadIdx.y < N && col < N)
            tileB[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads(); // wait for all threads to load tiles

        // multiply the two tiles
        for (int k = 0; k < TILE_SIZE; k++)
            sum += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];

        __syncthreads(); // wait for all threads before next tile
    }

    if (row < N && col < N)
        C[row * N + col] = sum;
}


extern "C" void gpu_matrix_multiply(float *h_A, float *h_B, float *h_C, int N) {
    size_t size = N * N * sizeof(float);
    
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // copy data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // define block and grid dimemsions
    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    matrixMultiplyTiled<<<grid, block>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    // check for any launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}