#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

__global__ void matrixMultiplyGPU(float *A, float *B, float *C, int N)
{

    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    // Each thread computes one element of the output matrix
    int row = blockIdx.y * TILE_SIZE + threadIdx.y; // global row index
    int col = blockIdx.x * TILE_SIZE + threadIdx.x; // global col index

    float sum = 0.0f;
    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; t++)
    {
         // load elements into shared memory if within bounds
        if (row < N && t * TILE_SIZE + threadIdx.x < N)
            tileA[threadIdx.y][threadIdx.x] = A[row * N + t * TILE_SIZE + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0f;

        if (t * TILE_SIZE + threadIdx.y < N && col < N)
            tileB[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads(); // wait for all threads to load tiles

        // multiply the two tiles
        for (int k = 0; k < TILE_SIZE; k++)
            sum += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];

        __syncthreads(); // wait for all threads before next tile
    }

    if (row < N && col < N)
        C[row * N + col] = sum;
}

void matrixMultiplyCPU(float *A, float *B, float *C, int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            float sum = 0.0f;
            for (int k = 0; k < N; k++)
            {
                sum += A[i * N + k] * B[k * N + j];
            }
            C[i * N + j] = sum;
        }
    }
}
int main(int argc, char **argv)
{
    int N = (argc > 1) ? atoi(argv[1]) : 1024; // allow matrix size as input
    size_t size = N * N * sizeof(float);
    
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);
    float *h_C_cpu = (float *)malloc(size);

    for (int i = 0; i < N * N; i++)
    {
        h_A[i] = rand() % 100 / 100.0f;
        h_B[i] = rand() % 100 / 100.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // copy data to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // define block and grid dimemsions
    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    // create CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // launch kernel with proper timing
    hipEventRecord(start);  // start recording before launch

    matrixMultiplyGPU<<<grid, block>>>(d_A, d_B, d_C, N);

    // check for any launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }

    // stop event after all GPU work finishes
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // measure elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU execution time (N=%d): %.6f seconds\n", N, milliseconds / 1000.0f);

    hipEventDestroy(start);
    hipEventDestroy(stop);


    // copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    
     // Free memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_cpu);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}